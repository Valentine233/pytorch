
#include <hip/hip_runtime.h>
__device__ unsigned int mulhilo32(
    unsigned int a,
    unsigned int b,
    unsigned int* result_high) {
  *result_high = __umulhi(a, b);
  return a * b;
}

__device__ uint4 single_round(uint4 ctr, uint2 key) {
  constexpr unsigned long kPhiloxSA = 0xD2511F53;
  constexpr unsigned long kPhiloxSB = 0xCD9E8D57;
  unsigned int hi0;
  unsigned int hi1;
  unsigned int lo0 = mulhilo32(kPhiloxSA, ctr.x, &hi0);
  unsigned int lo1 = mulhilo32(kPhiloxSB, ctr.z, &hi1);
  uint4 ret = {hi1 ^ ctr.y ^ key.x, lo1, hi0 ^ ctr.w ^ key.y, lo0};
  return ret;
}

__device__ uint4 philox(
    unsigned long long seed,
    unsigned long long subsequence,
    unsigned long long offset) {
  constexpr unsigned long kPhilox10A = 0x9E3779B9;
  constexpr unsigned long kPhilox10B = 0xBB67AE85;
  uint2 key = {};
  key.x = (unsigned int)seed;
  key.y = (unsigned int)(seed >> 32);
  uint4 counter = make_uint4(0, 0, 0, 0);
  counter.x = (unsigned int)(offset);
  counter.y = (unsigned int)(offset >> 32);
  counter.z = (unsigned int)(subsequence);
  counter.w = (unsigned int)(subsequence >> 32);

  uint4 output = {};
  uint2 key_ = key;
  uint4 counter_ = counter;
  for (int i = 0; i < 9; i++) {
    counter_ = single_round(counter_, key_);
    key_.x += (kPhilox10A);
    key_.y += (kPhilox10B);
  }
  output = single_round(counter_, key_);
  return output;
}

__device__ float uniformf(unsigned int x) {
  constexpr float kRanInvM32 = 2.3283064e-10f; // Inverse of 2^32.
  return x * kRanInvM32;
}

__device__ double uniform(unsigned int x, unsigned int y) {
  constexpr double kRan2Pow53Inv = 1.1102230246251565e-16;
  const unsigned long long z =
      (unsigned long long)x ^ ((unsigned long long)y << (53 - 32));
  return z * kRan2Pow53Inv + (kRan2Pow53Inv / 2.0);
}

__device__ double randLike(const uint4 &rng_result, int rng_component) {
  return uniform((&rng_result.x)[rng_component * 2], (&rng_result.x)[rng_component * 2 + 1]);
}

__device__ float randLikef(const uint4 &rng_result, int rng_component) {
  return uniformf((&rng_result.x)[rng_component]);
}
